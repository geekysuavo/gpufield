#include "hip/hip_runtime.h"

/* shapes.cu: generation of useful wire shapes for gpufield.
 * Copyright (C) 2014 Bradley Worley.
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License,
 * or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the
 *
 *     Free Software Foundation, Inc.
 *     59 Temple Place, Suite 330
 *     Boston, MA 02111-1307 USA
 */

/* include the shapes header. */
#include "shapes.h"

/* shapes_checkdir: checks the direction string for validity.
 * @dir: the direction stirng to check.
 */
int shapes_checkdir (const char *dir) {
  /* check for a valid direction string. */
  if (!dir ||
      strlen (dir) != 2 ||
      (dir[0] != '+' && dir[0] != '-') ||
      (dir[1] != 'x' && dir[1] != 'y' && dir[1] != 'z'))
    return 0;

  /* otherwise... */
  return 1;
}

/* shapes_arc: adds an arc wire shape into a wire list.
 * @wires: the wire list to add the arc to.
 * @origin: the center of the arc.
 * @radius: the radius of the arc.
 * @t1: the starting angle of the arc.
 * @t2: the ending angle of the arc.
 * @n: the number of segments in the arc.
 * @dir: the normal axis of the arc.
 * @I: the current flowing through the arc.
 */
int shapes_arc (wirelist *wires, vec3 origin,
                float radius, float t1, float t2,
                unsigned int n, char *dir, float I) {
  /* declare required variables. */
  float t, v1, v2;
  unsigned int i;
  vec3 a, b;

  /* check the direction string. */
  if (!shapes_checkdir (dir)) return 0;

  /* begin at the origin. */
  a = b = origin;

  /* loop through the arc segments. */
  for (i = 0; i < n; i++) {
    /* compute the angle value. */
    t = t1 + (t2 - t1) * (((float) i) / ((float) (n - 1)));

    /* compute the helix coordinates in principal axis space. */
    v1 = radius * cos (t * M_PI / 180.0);
    v2 = radius * sin (t * M_PI / 180.0);

    /* begin at the origin. */
    b = origin;

    /* work based on the normal vector. */
    if (dir[1] == 'x') {
      /* run the arc about the x-axis. */
      b.y += v1;
      b.z += v2;
    }
    else if (dir[1] == 'y') {
      /* run the arc about the y-axis. */
      b.x += v2;
      b.z += v1;
    }
    else if (dir[1] == 'z') {
      /* run the arc about the z-axis. */
      b.x += v1;
      b.y += v2;
    }

    /* see what we should do. */
    if (i > 0) {
      /* add the current segment into the wire list. */
      wires_add (wires, a, b, I);
    }

    /* move the start point to the end point for the next segment. */
    a = b;
  }

  /* return successfully. */
  return 1;
}

/* shapes_circle: adds a circular wire shape into a wire list.
 * @wires: the wire list to add the circle to.
 * @origin: the center of the circle.
 * @radius: the radius of the circle.
 * @n: the number of segments in the circle.
 * @dir: the normal axis of the circle.
 * @I: the current flowing through the circle.
 */
int shapes_circle (wirelist *wires, vec3 origin,
                   float radius, unsigned int n, char *dir, float I) {
  /* use the arc subroutine. */
  return shapes_arc (wires, origin, radius, 0.0, 360.0, n, dir, I);
}

/* shapes_helix: adds a helical wire shape into a wire list.
 * @wires: the wire list to add the helix to.
 * @origin: the starting point of the helix.
 * @radius: the radius of the helix.
 * @pitch: the pitch of the helix.
 * @turns: the number of helical turns.
 * @n: the number of segments in the helix.
 * @dir: a string denoting the helix direction.
 * @I: the current that the helix will carry.
 */
int shapes_helix (wirelist *wires, vec3 origin,
                  float radius, float pitch, float turns,
                  unsigned int n, char *dir, float I) {
  /* declare required variables. */
  vec3 a = {0.0, 0.0, 0.0};
  vec3 b = {0.0, 0.0, 0.0};
  float t, v1, v2, v3;
  unsigned int i;

  /* check the direction string. */
  if (!shapes_checkdir (dir)) return 0;

  /* begin at the origin. */
  a = b = origin;

  /* loop through the segments of the helix. */
  for (i = 0; i < n; i++) {
    /* compute the parameter variable. */
    t = (2.0 * M_PI * turns) * (((float) i) / ((float) (n - 1)));

    /* compute the helix coordinates in principal axis space. */
    v1 = radius * cos (t);
    v2 = radius * sin (t);
    v3 = (pitch * t) / (2.0 * M_PI);

    /* reflect the helix along its principal axis if a negative direction
     * was requested.
     */
    if (dir[0] == '-')
      v3 = -v3;

    /* begin at the origin. */
    b = origin;

    /* add the coordinates into the origin based on the directionality. */
    if (dir[1] == 'x') {
      /* extend along +x or -x. */
      b.x += v3;
      b.y += v1;
      b.z += v2;
    }
    else if (dir[1] == 'y') {
      /* extend along +y or -y. */
      b.x += v2;
      b.y += v3;
      b.z += v1;
    }
    else if (dir[1] == 'z') {
      /* extend along +z or -z. */
      b.x += v1;
      b.y += v2;
      b.z += v3;
    }

    /* see what we should do. */
    if (i > 0) {
      /* add the current segment into the wire list. */
      wires_add (wires, a, b, I);
    }

    /* move the start point to the end point for the next segment. */
    a = b;
  }

  /* return successfully. */
  return 1;
}

/* shapes_helmholtz: adds a helmholtz wire shape into a wire list.
 * (this takes the same arguments as shapes_helix)
 */
int shapes_helmholtz (wirelist *wires, vec3 origin,
                      float radius, float pitch, float turns,
                      unsigned int n, char *dir, float I) {
  /* declare required variables. */
  vec3 off1, off2;
  float Radj, Iadj;

  /* check the direction string. */
  if (!shapes_checkdir (dir)) return 0;

  /* initialize the offsets. */
  off1 = origin;
  off2 = origin;

  /* define a more precise starting point for the helices. */
  Radj = (pitch * turns) / 2.0;

  /* set the current based on helmholtz directionality. */
  Iadj = I;
  if (dir[0] == '-')
    Iadj = -Iadj;

  /* determine the directionality of the helices. */
  if (dir[1] == 'x') {
    /* arrange the helices along x. */
    off1.x += radius - Radj;
    off2.x -= radius + Radj;
  }
  else if (dir[1] == 'y') {
    /* arrange the helices along y. */
    off1.y += radius - Radj;
    off2.y -= radius + Radj;
  }
  else if (dir[1] == 'z') {
    /* arrange the helices along z. */
    off1.z += radius - Radj;
    off2.z -= radius + Radj;
  }

  /* try to add the first helix. */
  if (!shapes_helix (wires, off1, radius, pitch, turns, n, dir, Iadj))
    return 0;

  /* try to add the second helix. */
  if (!shapes_helix (wires, off2, radius, pitch, turns, n, dir, Iadj))
    return 0;

  /* return successfully. */
  return 1;
}

/* shapes_maxwell: adds a maxwell wire shape into a wire list.
 * (this takes the same arguments as shapes_helix)
 */
int shapes_maxwell (wirelist *wires, vec3 origin,
                    float radius, float pitch, float turns,
                    unsigned int n, char *dir, float I) {
  /* declare required variables. */
  vec3 off1, off2, off3;
  float d12, R12, Radj, Iadj;

  /* check the direction string. */
  if (!shapes_checkdir (dir)) return 0;

  /* initialize the offsets. */
  off1 = origin;
  off2 = origin;
  off3 = origin;

  /* define a more precise starting point for the helices. */
  Radj = (pitch * turns) / 2.0;
  d12 = sqrt (3.0 / 7.0) * radius;
  R12 = sqrt (4.0 / 7.0) * radius;

  /* set the current based on helmholtz directionality. */
  Iadj = I;
  if (dir[0] == '-')
    Iadj = -Iadj;

  /* determine the directionality of the helices. */
  if (dir[1] == 'x') {
    /* arrange the helices along x. */
    off1.x += d12 - Radj;
    off2.x -= d12 + Radj;
    off3.x -= Radj;
  }
  else if (dir[1] == 'y') {
    /* arrange the helices along y. */
    off1.y += d12 - Radj;
    off2.y -= d12 + Radj;
    off3.y -= Radj;
  }
  else if (dir[1] == 'z') {
    /* arrange the helices along z. */
    off1.z += d12 - Radj;
    off2.z -= d12 + Radj;
    off3.z -= Radj;
  }

  /* try to add the first helix. */
  if (!shapes_helix (wires, off1, R12, pitch, turns, n, dir, Iadj))
    return 0;

  /* try to add the second helix. */
  if (!shapes_helix (wires, off2, R12, pitch, turns, n, dir, Iadj))
    return 0;

  /* try to add the third helix. */
  if (!shapes_helix (wires, off3, radius, pitch, turns, n, dir, Iadj))
    return 0;

  /* return successfully. */
  return 1;
}

/* shapes_golay: adds a golay wire shape into a wire list.
 * @wires: the wire list to add the golay coil to.
 * @origin: the center point of the coil arrangement.
 * @a: the z-distance between the two coil pairs.
 * @b: the total z-axis length of the arrangement.
 * @c: the total 'dir'-axis length of the arrangement.
 * @theta: the arc angle of the saddle coils.
 * @radius: the radius of the coil arrangement.
 * @pitch: the winding pitch of the coil arrangement..
 * @turns: the number of windings in the coil arrangement..
 * @n: the number of segments per arc in the arrangement..
 * @dir: a string denoting the gradient direction.
 * @I: the current that the coil will carry.
 */
int shapes_golay (wirelist *wires, vec3 origin,
                  float a, float b, float c, float theta,
                  float radius, float pitch, unsigned int turns,
                  unsigned int n, char *dir, float I) {
  /* declare required variables. */
  float z1, z2, z3, z4, t, t1, t2, t3, t4, d1, d2, dx, dy;
  vec3 off1, off2, A, B;
  unsigned int i;

  /* check the direction string. */
  if (!shapes_checkdir (dir) || dir[1] == 'z')
    return 0;

  /* compute the z-axis offsets. */
  z1 = origin.z - b / 2.0;
  z2 = origin.z - a / 2.0;
  z3 = origin.z + a / 2.0;
  z4 = origin.z + b / 2.0;

  /* loop through the turns. */
  for (i = 0, t = theta; i < turns; i++) {
    /* compute the x/y offsets. */
    d1 = radius * cos (t / 2.0 * M_PI / 180.0) + c / 2.0;
    d2 = radius * sin (t / 2.0 * M_PI / 180.0);

    /* initialize the offsets. */
    off1 = off2 = origin;

    /* compute the offsets based on the gradient direction. */
    if (dir[1] == 'x') {
      /* compute for an x-gradient. */
      off1.x += c / 2.0;
      off2.x -= c / 2.0;

      /* compute the arc angles as well. */
      t1 = 0.0 - t / 2.0;
      t2 = 0.0 + t / 2.0;
      t3 = 180.0 - t / 2.0;
      t4 = 180.0 + t / 2.0;

      /* compute the x and y values. */
      dx = d1;
      dy = d2;
    }
    else if (dir[1] == 'y') {
      /* compute for a y-gradient. */
      off1.y += c / 2.0;
      off2.y -= c / 2.0;

      /* compute the arc angles as well. */
      t1 = 90.0 - t / 2.0;
      t2 = 90.0 + t / 2.0;
      t3 = 270.0 - t / 2.0;
      t4 = 270.0 + t / 2.0;

      /* compute the x and y values. */
      dx = d2;
      dy = d1;
    }

    /* draw the arcs of the first ring. */
    off1.z = off2.z = z1;
    shapes_arc (wires, off1, radius, t1, t2, n, "+z", I);
    shapes_arc (wires, off2, radius, t4, t3, n, "+z", I);

    /* draw the arcs of the second ring. */
    off1.z = off2.z = z2;
    shapes_arc (wires, off2, radius, t3, t4, n, "+z", I);
    shapes_arc (wires, off1, radius, t2, t1, n, "+z", I);

    /* draw the arcs of the third ring. */
    off1.z = off2.z = z3;
    shapes_arc (wires, off2, radius, t3, t4, n, "+z", I);
    shapes_arc (wires, off1, radius, t2, t1, n, "+z", I);

    /* draw the arcs of the fourth ring. */
    off1.z = off2.z = z4;
    shapes_arc (wires, off1, radius, t1, t2, n, "+z", I);
    shapes_arc (wires, off2, radius, t4, t3, n, "+z", I);

    /* draw the lower first wire. */
    A = vector (dx, -dy, z2);
    B = vector (dx, -dy, z1);
    wires_add (wires, A, B, I);

    /* draw the lower second wire. */
    A = vector (dx, dy, z1);
    B = vector (dx, dy, z2);
    wires_add (wires, A, B, I);

    /* draw the lower third wire. */
    A = vector (-dx, dy, z1);
    B = vector (-dx, dy, z2);
    wires_add (wires, A, B, I);

    /* draw the lower fourth wire. */
    A = vector (-dx, -dy, z2);
    B = vector (-dx, -dy, z1);
    wires_add (wires, A, B, I);

    /* draw the upper first wire. */
    A = vector (dx, -dy, z3);
    B = vector (dx, -dy, z4);
    wires_add (wires, A, B, I);

    /* draw the upper second wire. */
    A = vector (dx, dy, z4);
    B = vector (dx, dy, z3);
    wires_add (wires, A, B, I);

    /* draw the upper third wire. */
    A = vector (-dx, dy, z4);
    B = vector (-dx, dy, z3);
    wires_add (wires, A, B, I);

    /* draw the upper fourth wire. */
    A = vector (-dx, -dy, z3);
    B = vector (-dx, -dy, z4);
    wires_add (wires, A, B, I);

    /* adjust the z-axis extents of the next turn. */
    z1 += pitch;
    z2 -= pitch;
    z3 += pitch;
    z4 -= pitch;

    /* adjust the angle of the next turn. */
    t -= ((pitch / radius) * (180.0 / M_PI));
  }

  /* return successfully. */
  return 1;
}

/* shapes_squarespiral: winds a square planar spiral curve into a wire list.
 * @wires: the wire list to add the curve to.
 * @origin: the center of the spiral.
 * @width: the width of the spiral.
 * @turns: the number of spiral turns.
 * @I: the spiral current.
 */
int shapes_squarespiral (wirelist *wires, vec3 origin,
                         float width, float pitch, unsigned int turns,
                         float I) {
  /* declare required variables. */
  unsigned int i;
  vec3 a, b;
  float w;

  /* initialize the points. */
  a = b = vector (origin.x + width / 2.0, origin.y + width / 2.0, origin.z);

  /* loop through the turns. */
  for (i = 0, w = width; i < turns; i++) {
    /* make the first line. */
    b.x -= w;
    wires_add (wires, a, b, I);
    a = b;

    /* make the second line. */
    b.y -= w;
    wires_add (wires, a, b, I);
    a = b;

    /* change the length of the next two movements. */
    w -= pitch;

    /* make the third line. */
    b.x += w;
    wires_add (wires, a, b, I);
    a = b;

    /* make the fourth line. */
    b.y += w;
    wires_add (wires, a, b, I);
    a = b;

    /* change the length of the next two movements. */
    w -= pitch;
  }

  /* return successfully. */
  return 1;
}

