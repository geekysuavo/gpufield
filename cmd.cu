#include "hip/hip_runtime.h"

/* cmd.cu: command-parsing functions for gpufield.
 * Copyright (C) 2014 Bradley Worley.
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License,
 * or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the
 *
 *     Free Software Foundation, Inc.
 *     59 Temple Place, Suite 330
 *     Boston, MA 02111-1307 USA
 */

/* include the cmd header. */
#include "cmd.h"

/* cmd_parse_metric: parses metric-suffixed floating point numbers.
 * @str: the string to parse.
 */
float cmd_parse_metric (const char *str) {
  /* declare required variables. */
  unsigned int i;
  float val;

  /* convert the first part of the response string to a number. */
  val = atof (str);

  /* loop until a suitable last character is found. */
  i = 0; while (str[i] != ' ' && str[i] != '\n' && str[i] != '\0') i++;

  /* see if the number was suffixed with a metric prefix. */
  switch (str[i - 1]) {
    /* exa. */
    case 'E':
      val *= 1e+18;
      break;

    /* peta. */
    case 'P':
      val *= 1e+15;
      break;

    /* tera. */
    case 'T':
      val *= 1e+12;
      break;

    /* giga. */
    case 'G':
      val *= 1e+09;
      break;

    /* mega. */
    case 'M':
      val *= 1e+06;
      break;

    /* kilo. */
    case 'k':
      val *= 1e+03;
      break;

    /* centi. */
    case 'c':
      val *= 1e-02;
      break;

    /* milli. */
    case 'm':
      val *= 1e-03;
      break;

    /* micro. */
    case 'u':
      val *= 1e-06;
      break;

    /* nano. */
    case 'n':
      val *= 1e-09;
      break;

    /* pico. */
    case 'p':
      val *= 1e-12;
      break;

    /* femto. */
    case 'f':
      val *= 1e-15;
      break;

    /* atto. */
    case 'a':
      val *= 1e-18;
      break;

    /* anything else... */
    default:
      break;
  }

  /* return the parsed value. */
  return val;
}

/* cmd_parse: parses a string for possible commands.
 * @str: the input string to parse.
 * @cmd: the command name to check.
 * @n: the number of arguments to parse.
 * @...: (type, pointer) pairs, one per argument.
 */
int cmd_parse (const char *str, const char *cmd, unsigned int n, ...) {
  /* declare required variables. */
  unsigned int i, arg_type;
  char *pstr, *poff;
  void *arg_ptr;
  va_list vl;

  /* set the current location pointer. */
  pstr = (char *) str;

  /* loop until we find a non-whitespace character. */
  while ((*pstr == ' ' || *pstr == '\n') && *pstr != '\0')
    pstr++;

  /* check the string against the expected command. */
  if (strlen (pstr) < strlen (cmd) || strncmp (pstr, cmd, strlen (cmd)))
    return 0;

  /* loop until we find a whitespace character. */
  while (*pstr != ' ' && *pstr != '\n' && *pstr != '\0')
    pstr++;

  /* start parsing the argument list. */
  va_start (vl, n);

  /* loop for every expected argument. */
  for (i = 0; i < n; i++) {
    /* loop until we find a non-whitespace character. */
    while ((*pstr == ' ' || *pstr == '\n') && *pstr != '\0')
      pstr++;

    /* check that we're not at the end. */
    if (!strlen (pstr)) {
      /* end parsing the argument list and return failure. */
      va_end (vl);
      return 0;
    }

    /* get the argument type and pointer. */
    arg_type = va_arg (vl, unsigned int);
    arg_ptr = va_arg (vl, void *);

    /* act based on the argument type. */
    switch (arg_type) {
      /* null */
      case ARG_NULL:
        continue;

      /* %d */
      case ARG_INT:
        *((int *) arg_ptr) = atoi (pstr);
        break;

      /* %u */
      case ARG_UINT:
        *((unsigned int *) arg_ptr) = atoi (pstr);
        break;

      /* %c */
      case ARG_CHAR:
        *((char *) arg_ptr) = *pstr;
        break;

      /* %[^ ] */
      case ARG_STR:
        /* initialize the offset pointer. */
        poff = pstr;

        /* loop until we find a whitespace character. */
        while (*poff != ' ' && *poff != '\n' && *poff != '\0')
          poff++;

        /* copy the string over. */
        strncpy ((char *) arg_ptr, pstr, poff - pstr);
        ((char *) arg_ptr)[poff - pstr] = '\0';

        /* break parsing. */
        break;

      /* %f, extended. */
      case ARG_FLT:
        *((float *) arg_ptr) = cmd_parse_metric (pstr);
        break;
    }

    /* loop until we find a whitespace character. */
    while (*pstr != ' ' && *pstr != '\n' && *pstr != '\0')
      pstr++;
  }

  /* end parsing the argument list. */
  va_end (vl);

  /* return success. */
  return 1;
}

/* cmd_interpret: main command interpretation function.
 * @line: the command string to interpret.
 */
int cmd_interpret (char *line) {
  /* @a: starting location of wires / origin of grids.
   * @b: ending location of wires.
   */
  static vec3 a, b;

  /* @i: wire currents.
   * @x: parsing storage for x-values.
   * @y: parsing storage for y-values.
   * @z: parsing storage for z-values.
   */
  static float i;
  float x, y, z;

  /* @width: grid first-dimension extents and square spiral width.
   * @height: grid second-dimension extents.
   * @m: grid first-dimension divisions.
   * @n: grid second-dimension divisions.
   * @dim: dimension mode for grids.
   */
  float width, height;
  unsigned int m, n;
  char dim;

  /* @radius: radius of solenoids, arcs, circles, etc.
   * @pitch: wire diameter or winding pitch for multiturn structures.
   */
  float radius, pitch;

  /* @t: angle of arc for golay coil structures.
   * @t1: starting angle for arc structures.
   * @t2: ending angle for arc structures.
   */
  float t, t1, t2;

  /* @ga: 'a'-parameter (saddle separation) for golay coils.
   * @gb: 'b'-parameter (total length) for golay coils.
   * @gc: 'c'-parameter (radial saddle separation) for golay coils.
   */
  static float ga, gb, gc;

  /* @uturns: integral turn count.
   * @fturns: non-integral turn count.
   */
  unsigned int uturns;
  float fturns;

  /* @dir: direction string for solenoids, arcs, circles, etc.
   * @fname: output filename, or blank for standard output.
   */
  static char fname[256];
  char dir[256];

  /* @verb: verbosity flag.
   */
  static int verb = 0;

  /* declare pointers to a wire list and a grid. */
  static wirelist *wires;
  static grid *G;

  /* initialize the wire list. */
  if (!wires)
    wires = wires_alloc ();

  /* strip the newline from the end of the line buffer string. */
  if (line[strlen (line) - 1] == '\n')
    line[strlen (line) - 1] = '\0';

  /* skip parsing of empty lines. */
  if (strlen (line) <= 0)
    return 1;

  /* skip parsing of commented lines. */
  if (strlen (line) > 0 && line[0] == '#')
    return 1;

  /* parse the line buffer for command content. */
  if (cmd_parse (line, CMD_CURRENT, 1, ARG_FLT, &i)) {
    /* print a log message. */
    if (verb) logf ("set I = %.2e", i);
  }
  else if (cmd_parse (line, CMD_FILE, 1, ARG_STR, fname)) {
    /* print a log message. */
    if (verb) logf ("set file = '%s'", fname);
  }
  else if (cmd_parse (line, CMD_NOFILE, 0)) {
    /* set the file string. */
    strcpy (fname, "");

    /* print a log message. */
    if (verb) logf ("set file => stdout");
  }
  else if (cmd_parse (line, CMD_MOVETO, 3,
                      ARG_FLT, &x,
                      ARG_FLT, &y,
                      ARG_FLT, &z)) {
    /* move both endpoints to the new location. */
    a = b = vector (x, y, z);
  }
  else if (cmd_parse (line, CMD_LINETO, 3,
                      ARG_FLT, &x,
                      ARG_FLT, &y,
                      ARG_FLT, &z)) {
    /* move the end vector to the new location. */
    b = vector (x, y, z);

    /* add a wire between the start and end points. */
    wires_add (wires, a, b, i);

    /* move the start vector to the new location. */
    a = b;
  }
  else if (cmd_parse (line, CMD_CIRCLE, 3,
                      ARG_STR,  dir,
                      ARG_FLT,  &radius,
                      ARG_UINT, &n)) {
    /* try to build a circle. */
    if (!shapes_circle (wires, a, radius, n, dir, i))
      logf ("failed to build circle");
  }
  else if (cmd_parse (line, CMD_ARC, 5,
                      ARG_STR,  dir,
                      ARG_FLT,  &radius,
                      ARG_FLT,  &t1,
                      ARG_FLT,  &t2,
                      ARG_UINT, &n)) {
    /* try to build an arc. */
    if (!shapes_arc (wires, a, radius, t1, t2, n, dir, i))
      logf ("failed to build arc");
  }
  else if (cmd_parse (line, CMD_SOLENOID, 5,
                      ARG_STR,  dir,
                      ARG_FLT,  &radius,
                      ARG_FLT,  &pitch,
                      ARG_FLT,  &fturns,
                      ARG_UINT, &n)) {
    /* try to build a helix. */
    if (!shapes_helix (wires, a, radius, pitch, fturns, n, dir, i))
      logf ("failed to build solenoid");
  }
  else if (cmd_parse (line, CMD_HELMHOLTZ, 5,
                      ARG_STR,  dir,
                      ARG_FLT,  &radius,
                      ARG_FLT,  &pitch,
                      ARG_FLT,  &fturns,
                      ARG_UINT, &n)) {
    /* try to build a helmholtz arrangement. */
    if (!shapes_helmholtz (wires, a, radius, pitch, fturns, n, dir, i))
      logf ("failed to build helmholtz");
  }
  else if (cmd_parse (line, CMD_MAXWELL, 5,
                      ARG_STR,  dir,
                      ARG_FLT,  &radius,
                      ARG_FLT,  &pitch,
                      ARG_FLT,  &fturns,
                      ARG_UINT, &n)) {
    /* try to build a maxwell arrangement. */
    if (!shapes_maxwell (wires, a, radius, pitch, fturns, n, dir, i))
      logf ("failed to build maxwell");
  }
  else if (cmd_parse (line, CMD_GOLAY, 9,
                      ARG_STR,  dir,
                      ARG_FLT,  &ga,
                      ARG_FLT,  &gb,
                      ARG_FLT,  &gc,
                      ARG_FLT,  &t,
                      ARG_FLT,  &radius,
                      ARG_FLT,  &pitch,
                      ARG_UINT, &uturns,
                      ARG_UINT, &n)) {
    /* try to build a golay arrangement. */
    if (!shapes_golay (wires, a, ga, gb, gc, t,
                       radius, pitch, uturns,
                       n, dir, i))
      logf ("failed to build golay");
  }
  else if (cmd_parse (line, CMD_SQSPIRAL, 3,
                      ARG_FLT,  &width,
                      ARG_FLT,  &pitch,
                      ARG_UINT, &uturns)) {
    /* try to build a square planar spiral. */
    if (!shapes_squarespiral (wires, a, width, pitch, uturns, i))
      logf ("failed to build square planar spiral");
  }
  else if (cmd_parse (line, CMD_TRAJ, 4,
                      ARG_FLT,  &x,
                      ARG_FLT,  &y,
                      ARG_FLT,  &z,
                      ARG_UINT, &n)) {
    /* set the trajectory end point. */
    b = vector (x, y, z);

    /* check if verbose logging is enabled. */
    if (verb) {
      /* print a log message. */
      logf ("set A = { %.2e, %.2e, %.2e }", a.x, a.y, a.z);
      logf ("set B = { %.2e, %.2e, %.2e }", b.x, b.y, b.z);
      logf ("sample trajectory ( A -> B )");
    }

    /* allocate a grid and compute the field at its points. */
    G = grid_alloc_segment (n, a, b, wires);

    /* ensure the grid was successfully built. */
    if (!G) {
      /* nope. print a warning message and move along. */
      logf ("failed to compute field at grid coordinates");
      return 0;
    }

    /* write a log message. */
    if (verb) {
      if (strcmp (fname, "") == 0)
        logf ("write grid => stdout");
      else
        logf ("write grid => '%s'", fname);
    }

    /* try to write the grid to a file. */
    if (!grid_write (G, fname)) {
      /* failure. print a warning message and move along. */
      logf ("failed to write grid to '%s'", fname);
    }

    /* free the grid. */
    grid_free (G);

    /* set the new starting point. */
    a = b;
  }
  else if (cmd_parse (line, CMD_GRID, 5,
                      ARG_CHAR, &dim,
                      ARG_FLT,  &width,
                      ARG_FLT,  &height,
                      ARG_UINT, &m,
                      ARG_UINT, &n)) {
    /* ensure the dimension mode is valid. */
    if (dim != 'x' && dim != 'y' && dim != 'z') {
      /* invalid mode. print a warning message and move along. */
      logf ("invalid dimension '%c'", dim);
      return 0;
    }

    /* check if verbose logging is enabled. */
    if (verb) {
      /* yes, it is. print a log message. */
      logf ("set A = { %.2e, %.2e, %.2e }", a.x, a.y, a.z);
      logf ("sample ( d%c: %.2e, d%c: %.2e )",
            dim == 'x' ? 'y' : dim == 'y' ? 'x' : 'x', width,
            dim == 'x' ? 'z' : dim == 'y' ? 'z' : 'y', height);
    }

    /* allocate a grid and compute the field at its points. */
    G = grid_alloc_surface (m, n, a, width, height, dim, wires);

    /* ensure the grid was successfully built. */
    if (!G) {
      /* nope. print a warning message and move along. */
      logf ("failed to compute field at grid coordinates");
      return 0;
    }

    /* write a log message. */
    if (verb) {
      if (strcmp (fname, "") == 0)
        logf ("write grid => stdout");
      else
        logf ("write grid => '%s'", fname);
    }

    /* try to write the grid to a file. */
    if (!grid_write (G, fname)) {
      /* failure. print a warning message and move along. */
      logf ("failed to write grid to '%s'", fname);
    }

    /* free the grid. */
    grid_free (G);
  }
  else if (cmd_parse (line, CMD_WIRES, 0)) {
    /* print a log message. */
    if (verb) {
      if (strcmp (fname, "") == 0)
        logf ("write wires => stdout");
      else
        logf ("write wires => '%s'", fname);
    }

    /* try to write the wires to a file. */
    if (!wires_write (wires, fname)) {
      /* failure. print a warning message and move along. */
      logf ("failed to write wires to '%s'", fname);
    }
  }
  else if (cmd_parse (line, CMD_CLEAR, 0)) {
    /* print a log message. */
    if (verb) logf ("clearing wire list");

    /* free and re-initialize the wire list. */
    wires_free (wires);
    wires = wires_alloc ();

    /* ensure the wire list was re-initialized successfully. */
    if (!wires) {
      /* print an error message and exit. */
      logf ("failed to clear wire list");
      return 0;
    }

    /* reset the parsing state variables. */
    a = b = vector (0.0, 0.0, 0.0);
    i = 0.0;
  }
  else if (cmd_parse (line, CMD_END, 0)) {
    /* break the parsing loop to end execution. */
    if (verb) logf ("exiting...");
    return -1;
  }
  else if (cmd_parse (line, CMD_VERBOSE, 0)) {
    /* enable verbose logging. */
    logf ("verbose logging => on");
    verb = 1;
  }
  else if (cmd_parse (line, CMD_QUIET, 0)) {
    /* print a message. */
    if (verb) logf ("verbose logging => off");

    /* disable verbose logging. */
    verb = 0;
  }
  else {
    /* print a warning message. */
    logf ("unrecognized command '%s'", line);
  }

  /* return success. */
  return 1;
}

