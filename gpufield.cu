#include "hip/hip_runtime.h"

/* gpufield.cu: GPU-accelerated magnetostatics calculations via Biot Savart.
 * Copyright (C) 2014 Bradley Worley.
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License,
 * or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the
 *
 *     Free Software Foundation, Inc.
 *     59 Temple Place, Suite 330
 *     Boston, MA 02111-1307 USA
 */

/* include the gpufield header. */
#include "gpufield.h"

/* main: application entry point.
 * @argc: number of command-line arguments.
 * @argv: command-line argument string array.
 */
int main (int argc, char **argv) {
  /* declare required variables. */
  char buf[256], prompt[32], *pbuf;
  unsigned int kcmd;
  FILE *fh;
  int ret;

  /* see if the user passed a script filename on the command-line. */
  if (argc == 2) {
    /* open the input file. */
    fh = fopen (argv[1], "rb");
    if (!fh) {
      /* error out. */
      logf ("failed to open file '%s'", argv[1]);
      return 1;
    }

    /* loop while lines are available. */
    while (fgets (buf, sizeof (buf), fh)) {
      /* interpret the line. */
      ret = cmd_interpret (buf);

      /* see if execution was successful. */
      if (ret <= 0) {
        fclose (fh);
        return 1;
      }
    }

    /* close the input file. */
    fclose (fh);
  }
  else if (isatty (fileno (stdin))) {
    /* initialize the command counter. */
    kcmd = 1;

    /* loop until the session ends. */
    while (1) {
      /* build the prompt string. */
      snprintf (prompt, 32, "gpufield:%u> ", kcmd);
      pbuf = readline (prompt);

      /* end the session if no more input is available. */
      if (!pbuf)
        break;

      /* add the statement to the command history. */
      add_history (pbuf);

      /* interpret the line. */
      ret = cmd_interpret (pbuf);

      /* see if execution was successful. */
      if (ret < 0)
        break;

      /* increment the command counter for real commands. */
      if (strcmp (pbuf, ""))
        kcmd++;
    }
  }
  else {
    /* standard-input script mode. loop while lines are available. */
    while (fgets (buf, sizeof (buf), stdin)) {
      /* interpret the line. */
      ret = cmd_interpret (buf);

      /* see if execution was successful. */
      if (ret == 0)
        return 1;
      else if (ret < 0)
        break;
    }
  }

  /* return successfully. */
  return 0;
}

