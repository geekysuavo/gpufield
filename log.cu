#include "hip/hip_runtime.h"

/* log.cu: logging functions for gpufield.
 * Copyright (C) 2014 Bradley Worley.
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License,
 * or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the
 *
 *     Free Software Foundation, Inc.
 *     59 Temple Place, Suite 330
 *     Boston, MA 02111-1307 USA
 */

/* include the log header. */
#include "log.h"

/* define a convenience function for verbose logging. */
#define logf(...) logf_fn (__FILE__, __LINE__, __VA_ARGS__)

/* predeclare the core verbose logging function. */
void logf_fn (const char *f, const unsigned int l, const char *format, ...)
  __attribute__ ((format (printf, 3, 4)));

/* logf_fn: verbose logging core function.
 * @f: filename string.
 * @l: line number.
 * @format: printf-style format string.
 * @...: variable-length argument list.
 */
void logf_fn (const char *f, const unsigned int l, const char *format, ...) {
  /* declare the variable-length argument list. */
  va_list vl;

  /* print the first portion of the log string. */
  fprintf (stderr, "%s[%u]: ", f, l);

  /* print the user-controlled portion of the log string. */
  va_start (vl, format);
  vfprintf (stderr, format, vl);
  va_end (vl);

  /* tack on a newline. */
  fprintf (stderr, "\n");
  fflush (stderr);
}

