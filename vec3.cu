#include "hip/hip_runtime.h"

/* vec3.cu: three-dimensional vector types for gpufield.
 * Copyright (C) 2014 Bradley Worley.
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License,
 * or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the
 *
 *     Free Software Foundation, Inc.
 *     59 Temple Place, Suite 330
 *     Boston, MA 02111-1307 USA
 */

/* include the vec3 header. */
#include "vec3.h"

/* define the permittivity of vacuum. */
#define MU_0 (4.0 * M_PI * 1.0e-7)

/* vector: create a 3-vector.
 * @x: the x-component.
 * @y: the y-component.
 * @z: the z-component.
 */
vec3 vector (float x, float y, float z) {
  /* declare the output vector. */
  vec3 v;

  /* set the vector components. */
  v.x = x;
  v.y = y;
  v.z = z;

  /* return the vector. */
  return v;
}

/* vcmp: compare two 3-vectors.
 * @a: the first vector.
 * @b: the second vector.
 */
int vcmp (vec3 a, vec3 b) {
  /* compute and return equality. */
  return ((a.x == b.x) && (a.y == b.y) && (a.z == b.z));
}

/* len: compute the length (euclidean norm) of a 3-vector.
 * @v: the vector to compute.
 */
float len (vec3 v) {
  /* compute and return the length. */
  return sqrt (v.x * v.x + v.y * v.y + v.z * v.z);
}

/* dot: compute the dot product of two 3-vectors.
 * @a: the first vector in the product.
 * @b: the second vector in the product.
 */
float dot (vec3 a, vec3 b) {
  /* compute and return the dot product. */
  return (a.x * b.x + a.y * b.y + a.z * b.z);
}

/* cross: compute the cross product of two 3-vectors.
 * @a: the first vector in the product.
 * @b: the second vector in the product.
 */
vec3 cross (vec3 a, vec3 b) {
  /* declare the output vector. */
  vec3 c;

  /* compute the elements of the output vector. */
  c.x = a.y * b.z - a.z * b.y;
  c.y = a.z * b.x - a.x * b.z;
  c.z = a.x * b.y - a.y * b.x;

  /* return the output vector. */
  return c;
}

/* unit: compute the unit 3-vector of a 3-vector.
 * @v: the vector to compute.
 */
vec3 unit (vec3 v) {
  /* declare the output vector. */
  float l;
  vec3 u;

  /* compute the input vector length. */
  l = len (v);

  /* compute the components of the output vector. */
  u.x = v.x / l;
  u.y = v.y / l;
  u.z = v.z / l;

  /* return the output vector. */
  return u;
}

/* scale: scale a 3-vector by a scalar value.
 * @alpha: the scalar value.
 * @v: the vector to scale.
 */
vec3 scale (float alpha, vec3 v) {
  /* declare the output vector. */
  vec3 s;

  /* compute the elements of the output vector. */
  s.x = alpha * v.x;
  s.y = alpha * v.y;
  s.z = alpha * v.z;

  /* return the output vector. */
  return s;
}

/* proj: project a 3-vector onto another unit 3-vector.
 * @v: the 3-vector to compute.
 * @u: the unit 3-vector to project @v onto.
 */
vec3 proj (vec3 v, vec3 u) {
  /* compute and return the projection. */
  return scale (dot (v, u), u);
}

/* add: add two 3-vectors.
 * @a: the first vector.
 * @b: the second vector.
 */
vec3 add (vec3 a, vec3 b) {
  /* declare the output vector. */
  vec3 c;

  /* compute the elements of the output vector. */
  c.x = a.x + b.x;
  c.y = a.y + b.y;
  c.z = a.z + b.z;

  /* return the output vector. */
  return c;
}

/* sub: subtract two 3-vectors.
 * @a: the first vector.
 * @b: the second vector.
 */
vec3 sub (vec3 a, vec3 b) {
  /* declare the output vector. */
  vec3 c;

  /* compute the elements of the output vector. */
  c.x = a.x - b.x;
  c.y = a.y - b.y;
  c.z = a.z - b.z;

  /* return the output vector. */
  return c;
}

/* vinterp: interpolates between two 3-vectors.
 * @a: the starting vector.
 * @b: the ending vector.
 * @t: the interpolation factor.
 */
vec3 vinterp (vec3 a, vec3 b, float t) {
  /* compute and return the output vector. */
  return add (a, scale (t, sub (b, a)));
}

/* field: compute the magnetic field vector at a point M due to an
 * infinitely narrow wire along segment AB carrying current I.
 */
vec3 field (vec3 A, vec3 B, vec3 M, float I) {
  /* declare required variables. */
  float c1, c2, dLM, mag;
  vec3 vAB, vAM, vBM, vLM;
  vec3 uAB, uAM, uBM, uLM;
  vec3 f;

  /* compute the vector between the endpoints of the wire. */
  vAB = sub (B, A);
  uAB = unit (vAB);

  /* compute the vector from the start point to the calculation point. */
  vAM = sub (M, A);
  uAM = unit (vAM);

  /* compute the vector from the end point to the calculation point. */
  vBM = sub (M, B);
  uBM = unit (vBM);

  /* find the angles from the wire ends to the calculation point. */
  c1 = dot (uAB, uAM);
  c2 = dot (uAB, uBM);

  /* compute a vector from the wire to the calculation point, such that
   * the vector is perpindicular to the wire.
   */
  vLM = sub (vAM, proj (vAM, uAB));
  uLM = unit (vLM);
  dLM = len (vLM);

  /* compute the magnitude of the field vector. */
  mag = ((MU_0 * I) / (4.0 * M_PI * dLM)) * (c1 - c2);

  /* compute the direction (and scale it) of the field vector. */
  f = cross (uAB, uLM);
  f = scale (mag, f);

  /* return the computed field value. */
  return f;
}

