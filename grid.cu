#include "hip/hip_runtime.h"

/* grid.cu: gridded sampling of fields for gpufield.
 * Copyright (C) 2014 Bradley Worley.
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License,
 * or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the
 *
 *     Free Software Foundation, Inc.
 *     59 Temple Place, Suite 330
 *     Boston, MA 02111-1307 USA
 */

/* include the grid header. */
#include "grid.h"

/* fval: macro to extract the field vector from a grid in row-major form.
 * @G: the grid pointer to operate on.
 * @i: the row index.
 * @j: the column index.
 */
#define fval(G, i, j) ((G)->f[(i) * (G)->n + (j)])

/* xyzval: macro to extract the location from a grid in row-major form.
 * @G: the grid pointer to operate on.
 * @i: the row index.
 * @j: the column index.
 */
#define xyzval(G, i, j) ((G)->xyz[(i) * (G)->n + (j)])

/* grid_exec_gpu_task: gpu kernel for computing the field at a grid point.
 * @a: array of wire starting points.
 * @b: array of wire ending points.
 * @I: array of wire currents.
 * @g: array of grid coordinates.
 * @f: array of field values.
 * @J: number of wires.
 */
__global__ void grid_exec_gpu_task (float *a, float *b, float *I,
                                    float *g, float *f, int J) {
  /* declare all required intermediate variables. */
  float ax, ay, az, bx, by, bz, gx, gy, gz, fx, fy, fz;
  float vABx, vABy, vABz, uABx, uABy, uABz, dAB;
  float vAMx, vAMy, vAMz, uAMx, uAMy, uAMz, dAM;
  float vBMx, vBMy, vBMz, uBMx, uBMy, uBMz, dBM;
  float vLMx, vLMy, vLMz, uLMx, uLMy, uLMz, dLM;
  float vALx, vALy, vALz;
  float c1, c2, prj, c;
  float current;

  /* get the kernel index and declare a wire loop counter. */
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j;

  /* get the current grid coordinates. */
  gx = g[3 * i];
  gy = g[3 * i + 1];
  gz = g[3 * i + 2];

  /* initialize the field value. */
  fx = fy = fz = 0.0;

  /* loop for all wires. */
  for (j = 0; j < J; j++) {
    /* get the wire starting point. */
    ax = a[3 * j];
    ay = a[3 * j + 1];
    az = a[3 * j + 2];

    /* get the wire ending point. */
    bx = b[3 * j];
    by = b[3 * j + 1];
    bz = b[3 * j + 2];

    /* get the wire current. */
    current = I[3 * j];

    /* compute the vector between the wire ends. */
    vABx = bx - ax;
    vABy = by - ay;
    vABz = bz - az;

    /* compute the vector between the start and the grid. */
    vAMx = gx - ax;
    vAMy = gy - ay;
    vAMz = gz - az;

    /* compute the vector between the end and the grid. */
    vBMx = gx - bx;
    vBMy = gy - by;
    vBMz = gz - bz;

    /* compute the lengths of AB, AM, and BM. */
    dAB = sqrt (vABx * vABx + vABy * vABy + vABz * vABz);
    dAM = sqrt (vAMx * vAMx + vAMy * vAMy + vAMz * vAMz);
    dBM = sqrt (vBMx * vBMx + vBMy * vBMy + vBMz * vBMz);

    /* compute the unit vector between the wire ends. */
    uABx = vABx / dAB;
    uABy = vABy / dAB;
    uABz = vABz / dAB;

    /* compute the unit vector between the start and the grid. */
    uAMx = vAMx / dAM;
    uAMy = vAMy / dAM;
    uAMz = vAMz / dAM;

    /* compute the unit vector between the end and the grid. */
    uBMx = vBMx / dBM;
    uBMy = vBMy / dBM;
    uBMz = vBMz / dBM;

    /* find the angles from the wire ends to the grid point. */
    c1 = uABx * uAMx + uABy * uAMy + uABz * uAMz;
    c2 = uABx * uBMx + uABy * uBMy + uABz * uBMz;

    /* compute the dot product of AM and AB. */
    prj = vAMx * uABx + vAMy * uABy + vAMz * uABz;

    /* project AM onto AB to yield AL. */
    vALx = uABx * prj;
    vALy = uABy * prj;
    vALz = uABz * prj;

    /* compute the vector between the wire inner point and the grid. */
    vLMx = vAMx - vALx;
    vLMy = vAMy - vALy;
    vLMz = vAMz - vALz;

    /* compute the length of LM. */
    dLM = sqrt (vLMx * vLMx + vLMy * vLMy + vLMz * vLMz);

    /* compute the unit vector between the wire inner point and the grid. */
    uLMx = vLMx / dLM;
    uLMy = vLMy / dLM;
    uLMz = vLMz / dLM;

    /* compute the magnetic field unit vector components. */
    fx = uABy * uLMz - uABz * uLMy;
    fy = uABz * uLMx - uABx * uLMz;
    fz = uABx * uLMy - uABy * uLMx;

    /* compute the magnetic field magnitude and scale
     * the vector components accordingly.
     */
    c = 1.0e-7f * current * (c1 - c2) / dLM;
    fx *= c;
    fy *= c;
    fz *= c;

    /* sum in the current wire's contribution to the field. */
    f[3 * i + 0] += fx;
    f[3 * i + 1] += fy;
    f[3 * i + 2] += fz;
  }
}

/* grid_exec_gpu: computes the gridded field values using the gpu.
 * @G: the grid to compute field values for.
 * @wires: the wire list.
 */
void grid_exec_gpu (grid *G, wirelist *wires) {
  /* declare host arrays. */
  float *wmem, *gmem;
  
  /* declare device arrays. */
  float *a, *b, *i, *g, *f;

  /* declare array size variables. */
  unsigned int nw, ng, m, n;

  /* declare a loop index. */
  unsigned int k;

  /* compute the array sizes. */
  nw = 3 * wires->n;
  ng = 3 * G->m * G->n;

  /* allocate the host arrays. */
  wmem = (float *) calloc (nw, sizeof (float));
  gmem = (float *) calloc (ng, sizeof (float));

  /* fill the nw-length host array with information for 'a'. */
  for (k = 0; k < wires->n; k++) {
    wmem[3 * k + 0] = wires->A[k].x;
    wmem[3 * k + 1] = wires->A[k].y;
    wmem[3 * k + 2] = wires->A[k].z;
  }

  /* allocate the device 'a' array and copy the host array into it. */
  hipMalloc ((void **) &a, nw * sizeof (float));
  hipMemcpy (a, wmem, nw * sizeof (float), hipMemcpyHostToDevice);

  /* fill the nw-length host array with information for 'b'. */
  for (k = 0; k < wires->n; k++) {
    wmem[3 * k + 0] = wires->B[k].x;
    wmem[3 * k + 1] = wires->B[k].y;
    wmem[3 * k + 2] = wires->B[k].z;
  }

  /* allocate the device 'b' array and copy the host array into it. */
  hipMalloc ((void **) &b, nw * sizeof (float));
  hipMemcpy (b, wmem, nw * sizeof (float), hipMemcpyHostToDevice);

  /* fill the nw-length host array with information for 'i'. */
  for (k = 0; k < wires->n; k++) {
    wmem[3 * k + 0] = wires->i[k];
    wmem[3 * k + 1] = wires->i[k];
    wmem[3 * k + 2] = wires->i[k];
  }

  /* allocate the device 'i' array and copy the host array into it. */
  hipMalloc ((void **) &i, nw * sizeof (float));
  hipMemcpy (i, wmem, nw * sizeof (float), hipMemcpyHostToDevice);

  /* fill the ng-length host array with information for 'g'. */
  for (k = 0; k < G->m * G->n; k++) {
    gmem[3 * k + 0] = G->xyz[k].x;
    gmem[3 * k + 1] = G->xyz[k].y;
    gmem[3 * k + 2] = G->xyz[k].z;
  }

  /* allocate the device 'g' array and copy the host array into it. */
  hipMalloc ((void **) &g, ng * sizeof (float));
  hipMemcpy (g, gmem, ng * sizeof (float), hipMemcpyHostToDevice);

  /* fill the ng-length host array with information for 'f'. */
  for (k = 0; k < ng; k++)
    gmem[k] = 0.0;

  /* allocate the device 'f' array and copy the host array into it. */
  hipMalloc ((void **) &f, ng * sizeof (float));
  hipMemcpy (f, gmem, ng * sizeof (float), hipMemcpyHostToDevice);

  /* initialize the kernel block and thread sizes. */
  m = G->m;
  n = G->n;

  /* check if we can fix up the thread size. */
  while (n > 1024) {
    m *= 2;
    n /= 2;
  }

  /* execute the gpu kernel. */
  grid_exec_gpu_task<<<m, n>>> (a, b, i, g, f, wires->n);

  /* copy back the result from the gpu into the ng-length host array. */
  hipMemcpy (gmem, f, ng * sizeof (float), hipMemcpyDeviceToHost);

  /* extract back information from the ng-length host array
   * into the grid structure.
   */
  for (k = 0; k < G->m * G->n; k++) {
    G->f[k].x = gmem[3 * k + 0];
    G->f[k].y = gmem[3 * k + 1];
    G->f[k].z = gmem[3 * k + 2];
  }

  /* free the device arrays. */
  hipFree (a);
  hipFree (b);
  hipFree (i);
  hipFree (g);
  hipFree (f);

  /* free the host arrays. */
  free (wmem);
  free (gmem);
}

/* grid_exec_cpu: computes the gridded field values using a single cpu core.
 * @G: the grid to compute field values for.
 * @wires: the wire list.
 */
void grid_exec_cpu (grid *G, wirelist *wires) {
  /* declare looping variables. */
  unsigned int i, j;

  /* loop over the grid array index. */
  for (i = 0; i < G->m * G->n; i++) {
    /* loop over the wire list array index. */
    for (j = 0; j < wires->n; j++) {
      /* add in the j-th wire's contribution to the i-th grid point. */
      G->f[i] = add (G->f[i], field (wires->A[j], wires->B[j], G->xyz[i],
                                     wires->i[j]));
    }
  }
}

/* grid_alloc: allocates a certain size grid.
 * @m: the number of first-dim points.
 * @n: the number of second-dim points.
 */
grid *grid_alloc (unsigned int m, unsigned int n) {
  /* allocate the grid pointer. */
  grid *G = (grid *) malloc (sizeof (grid));
  if (!G) return NULL;

  /* check the grid size. */
  if ((n & (n - 1)) != 0) {
    /* print an error message and return nothing. */
    logf ("grid second dimension must be a power of two");
    return NULL;
  }

  /* store the grid size into the pointer. */
  G->m = m;
  G->n = n;

  /* allocate the grid coordinates array. */
  G->xyz = (vec3 *) calloc (G->m * G->n, sizeof (vec3));
  if (!G->xyz) return NULL;

  /* allocate the field values array. */
  G->f = (vec3 *) calloc (G->m * G->n, sizeof (vec3));
  if (!G->f) return NULL;

  /* return the grid pointer. */
  return G;
}

/* grid_alloc_segment: allocates a grid segment and computes its values.
 * @n: the number of grid segment points.
 * @A: the starting point of the segment.
 * @B: the ending point of the segment.
 * @wires: the wire list to use during calculation.
 */
grid *grid_alloc_segment (unsigned int n, vec3 A, vec3 B,
                          wirelist *wires) {
  /* declare loop variables and the output grid pointer. */
  unsigned int i;
  grid *G;

  /* allocate the grid pointer. */
  G = grid_alloc (1, n);
  if (!G) return NULL;

  /* loop through the grid elements. */
  for (i = 0; i < n; i++) {
    /* interpolate between the two points. */
    G->xyz[i] = vinterp (A, B, ((float) i) / ((float) (n - 1)));
  }

  /* use the gpu to calculate the field values at the grid coordinates. */
  grid_exec_gpu (G, wires);

  /* return the allocated grid pointer. */
  return G;
}

/* grid_alloc_surface: allocates a grid surface and computes its values.
 * @m: the number of first-dimension grid points.
 * @n: the number of second-dimension grid points.
 * @origin: the origin of the grid.
 * @u: the first-dimension extents.
 * @v: the second-dimension extents.
 * @dim: the dimension to keep constant.
 * @wires: the wire list to use during calculation.
 */
grid *grid_alloc_surface (unsigned int m, unsigned int n,
                          vec3 origin, float u, float v,
                          char dim, wirelist *wires) {
  /* declare loop variables and the output grid pointer. */
  unsigned int i, j;
  grid *G;

  /* allocate the grid pointer. */
  G = grid_alloc (m, n);
  if (!G) return NULL;

  /* determine the grid constant dimension. */
  if (dim == 'x' || dim == 'X') {
    /* loop through the y-dimension grid points. */
    for (i = 0; i < m; i++) {
      /* loop through the z-dimension grid points. */
      for (j = 0; j < n; j++) {
        /* compute the grid coordinate at (i,j). */
        xyzval (G, i, j) = origin;
        xyzval (G, i, j).y += u * ((float) i / (float) m) - (u / 2.0);
        xyzval (G, i, j).z += v * ((float) j / (float) n) - (v / 2.0);
      }
    }
  }
  else if (dim == 'y' || dim == 'Y') {
    /* loop through the x-dimension grid points. */
    for (i = 0; i < m; i++) {
      /* loop through the z-dimension grid points. */
      for (j = 0; j < n; j++) {
        /* compute the grid coordinate at (i,j). */
        xyzval (G, i, j) = origin;
        xyzval (G, i, j).x += u * ((float) i / (float) m) - (u / 2.0);
        xyzval (G, i, j).z += v * ((float) j / (float) n) - (v / 2.0);
      }
    }
  }
  else if (dim == 'z' || dim == 'Z') {
    /* loop through the x-dimension grid points. */
    for (i = 0; i < m; i++) {
      /* loop through the y-dimension grid points. */
      for (j = 0; j < n; j++) {
        /* compute the grid coordinate at (i,j). */
        xyzval (G, i, j) = origin;
        xyzval (G, i, j).x += u * ((float) i / (float) m) - (u / 2.0);
        xyzval (G, i, j).y += v * ((float) j / (float) n) - (v / 2.0);
      }
    }
  }
  else {
    /* invalid constant dimension. */
    return NULL;
  }

  /* use the gpu to calculate the field values at the grid coordinates. */
  grid_exec_gpu (G, wires);

  /* return the allocated grid pointer. */
  return G;
}

/* grid_free: frees an allocated grid pointer.
 * @G: the grid pointer to free.
 */
void grid_free (grid *G) {
  /* don't free a null pointer. */
  if (!G) return;

  /* check if there are grid elements to free. */
  if (G->m > 0 || G->n > 0) {
    /* free the grid coordinate and field arrays. */
    free (G->xyz);
    free (G->f);

    /* set the grid sizes to zero. */
    G->m = 0;
    G->n = 0;
  }

  /* free the grid pointer. */
  free (G);
}

/* grid_write: writes a grid's data to a text-format file.
 * @G: the grid to extract data from.
 * @filename: the output filename.
 */
int grid_write (grid *G, const char *filename) {
  /* declare output variables. */
  unsigned int i;
  vec3 xyz, f;
  FILE *fh;

  /* check if an actual filename was passed. */
  if (strcmp (filename, "")) {
    /* yes. open the output file for writing. */
    fh = fopen (filename, "wb");
    if (!fh) return 0;
  }

  /* loop through the grid array elements. */
  for (i = 0; i < G->m * G->n; i++) {
    /* extract the grid coordinate and field value. */
    xyz = G->xyz[i];
    f = G->f[i];

    /* print the coordinate and field value to the output file. */
    if (strcmp (filename, "")) {
      /* print to the output file handle. */
      fprintf (fh, "%u %e %e %e %e %e %e %e\n", i,
               xyz.x, xyz.y, xyz.z,
               f.x, f.y, f.z,
               len (f));
     }
     else {
      /* print to standard output. */
      fprintf (stdout, "%u %e %e %e %e %e %e %e\n", i,
               xyz.x, xyz.y, xyz.z,
               f.x, f.y, f.z,
               len (f));
    }
  }

  /* close the output file. */
  if (strcmp (filename, ""))
    fclose (fh);

  /* return success. */
  return 1;
}

